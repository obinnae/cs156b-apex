void gradient(const float * const * u, 
                 const float * const * v,
                 const int index,
                 const float * y
                 Baseline *b,
                 const int factor_length,
                 float lambda,
                 bool isU
                 int k,
                 int sizeofV){
    /* k should be num factors, sizeofV is number of movies or users */
    if(isU){
        hipMemcpy(dev_U, u[index], k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_V, v, sizeofV * k * sizeof(float), hipMemcpyHostToDevice);
        
    }else{
        hipMemcpy(dev_U, v[index], k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_V, u, sizeofV * k * sizeof(float), hipMemcpyHostToDevice);
       
    }
    hipMemcpy(dev_Y, y, sizeofV * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_lambda, lambda, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_sizeofV, sizeofV, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_K, k, sizeof(int), hipMemcpyHostToDevice);
    cudaGradient<<<1024, 32>>> (dev_U, dev_V, dev_Y, dev_lambda, dev_sizeofV, dev_K);
    if(isU){
        hipMemcpy(u[index], dev_U, k * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(v, dev_V, sizeofV * k * sizeof(float), hipMemcpyDeviceToHost);
    }else{
        hipMemcpy(v[index], dev_U, k * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(u, dev_V, sizeofV * k * sizeof(float), hipMemcpyDeviceToHost);
    }
    float *sum = malloc(sizeofV * sizeof(float));
    hipMemcpy(sum, out, sizeofV * sizeof(float), hipMemcpyDeviceToHost);
    int i;
    float total = 0;
    for (i = 0; i < sizeofV; i++){
        total += sum[i];
    }

    float *grad = malloc(k * sizeof(float));
    for (i = 0; i < k; i++){
        grad[i] = lambda * dev_U[i] - dev_V[i];
    }
                         

                     
}

void cudaGradient(const float *dev_U, const float * const * dev_V, const float * dev_Y, float dev_lambda, int dev_sizeofV, int dev_K, const float *out){

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < dev_sizeofV){
        unsigned int i = 0;
        float sum = 0;
        for (i = 0; i < dev_K; i++){
            sum += dev_U[i] * dev_V[index][i];
        }
        sum = dev_Y[index] - sum;
    }
    out[index] = sum;


}
